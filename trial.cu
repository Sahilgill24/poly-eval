// ** Trial for my Personal Reference **

#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <cassert>
#include <cstdlib>
#include <cstdio>

// Similiar to Matrix Multiplication
// There we use a 2D grid and 2D blocks
// O(bcd), so can be parallelized
#define HOST_DEVICE_INLINE __host__ __device__ __forceinline__
#define HOST_INLINE __host__ __forceinline__

#define P_MOD 32843
template <unsigned P = P_MOD>

class Dummy_Scalar
{
public:
    static constexpr unsigned MODULUS = P;

    unsigned x;

    static HOST_DEVICE_INLINE Dummy_Scalar zero() { return {0}; }

    static HOST_DEVICE_INLINE Dummy_Scalar one() { return {1}; }

    friend HOST_INLINE std::ostream &operator<<(std::ostream &os, const Dummy_Scalar &scalar)
    {
        os << scalar.x;
        return os;
    }

    friend HOST_DEVICE_INLINE Dummy_Scalar operator+(Dummy_Scalar p1, const Dummy_Scalar &p2)
    {
        return {(p1.x + p2.x) % MODULUS};
    }

    friend HOST_DEVICE_INLINE Dummy_Scalar operator*(Dummy_Scalar p1, const Dummy_Scalar &p2)
    {
        return {(p1.x * p2.x) % MODULUS};
    }

    friend HOST_DEVICE_INLINE bool operator==(const Dummy_Scalar &p1, const Dummy_Scalar &p2)
    {
        return (p1.x == p2.x);
    }

    friend HOST_DEVICE_INLINE bool operator!=(const Dummy_Scalar &p1, const Dummy_Scalar &p2)
    {
        return (p1.x != p2.x);
    }

    friend HOST_DEVICE_INLINE bool operator==(const Dummy_Scalar &p1, const unsigned p2)
    {
        return (p1.x == p2);
    }

    static HOST_DEVICE_INLINE Dummy_Scalar neg(const Dummy_Scalar &scalar)
    {
        return {MODULUS - scalar.x};
    }

    static HOST_INLINE Dummy_Scalar rand_host()
    {
        return {static_cast<unsigned>(rand() % MODULUS)};
    }
};

typedef Dummy_Scalar<> test_scalar;

__global__ void horner_kernel(const test_scalar *coeffs,
                              const test_scalar *domain,
                              int coeffs_size,
                              int domain_size,
                              int batch_size,
                              test_scalar *evals)
{
    int idx_in_batch = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx_in_batch >= batch_size)
        return;

    const test_scalar *curr_coeffs = coeffs + idx_in_batch * coeffs_size;
    test_scalar *curr_evals = evals + idx_in_batch * domain_size;

    for (int eval_idx = 0; eval_idx < domain_size; ++eval_idx)
    {
        curr_evals[eval_idx] = curr_coeffs[coeffs_size - 1];
        for (int coeff_idx = coeffs_size - 2; coeff_idx >= 0; --coeff_idx)
        {
            curr_evals[eval_idx] =
                curr_evals[eval_idx] * domain[eval_idx] + curr_coeffs[coeff_idx];
        }
    }
}
// one includes the kernel without the polynomials being batched.
// for (uint64_t idx_in_batch = 0; idx_in_batch < batch_size; ++idx_in_batch)
// {
//     const Scalar *curr_coeffs = coeffs + idx_in_batch * coeffs_size;
//     Scalar *curr_evals = evals + idx_in_batch * domain_size;
//     for (uint64_t eval_idx = 0; eval_idx < domain_size; ++eval_idx)
//     {
//         curr_evals[eval_idx] = curr_coeffs[coeffs_size - 1];
//         for (int64_t coeff_idx = coeffs_size - 2; coeff_idx >= 0; --coeff_idx)
//         {
//             curr_evals[eval_idx] =
//                 curr_evals[eval_idx] * domain[eval_idx] + curr_coeffs[coeff_idx];
//         }
//     }
// }
__global__ void horner_kernel2() {}

void poly_eval(
    const test_scalar *coeffs,
    const test_scalar *domain,
    int coeffs_size,
    int domain_size,
    int batch_size,
    test_scalar *evals /*OUT*/)
{
    dim3 grid_size((batch_size + 255) / 256);
    dim3 block_size(256);
    horner_kernel<<<grid_size, block_size>>>(coeffs, domain, coeffs_size, domain_size, batch_size, evals);
}

void polyeval2(
    const test_scalar *coeffs,
    const test_scalar *domain,
    int coeffs_size,
    int domain_size,
    int batch_size,
    test_scalar *evals /*OUT*/)
{
    // using the kernel here
}

int main()
{
    hipEvent_t start, stop;
    float time;

    // test parameters
    int coeffs_log_size = 22;
    int coeffs_size = 1 << coeffs_log_size;
    int domain_size = 7;
    int batch_size = 10;
    int total_coeffs_size = batch_size * coeffs_size;
    int total_results_size = batch_size * domain_size;

    printf("running poly eval of degree, 2^%d, domain_size=%d, batch_size=%d, scalar modulus=%d\n", coeffs_log_size, domain_size, batch_size, P_MOD);

    // init inputs
    test_scalar *coeffs = new test_scalar[total_coeffs_size];
    test_scalar *domain = new test_scalar[domain_size];
    for (int i = 0; i < total_coeffs_size; i++)
    {
        coeffs[i] = test_scalar::rand_host();
    }
    for (int i = 0; i < domain_size; i++)
    {
        domain[i] = test_scalar::rand_host();
    }

    std::cout << "finished generating inputs" << std::endl;

    test_scalar *results = new test_scalar[total_results_size];
    test_scalar *ref_results = new test_scalar[total_results_size];

    // allocate memory and copy to device
    test_scalar *d_coeffs;
    test_scalar *d_domain;
    test_scalar *d_results;
    hipMalloc(&d_coeffs, sizeof(test_scalar) * total_coeffs_size);
    hipMalloc(&d_domain, sizeof(test_scalar) * domain_size);
    hipMalloc(&d_results, sizeof(test_scalar) * total_results_size);
    hipMemcpy(d_coeffs, coeffs, sizeof(test_scalar) * total_coeffs_size, hipMemcpyHostToDevice);
    hipMemcpy(d_domain, domain, sizeof(test_scalar) * domain_size, hipMemcpyHostToDevice);

    std::cout << "finished copying to device" << std::endl;

    hipEventCreate(&start);
    hipEventCreate(&stop);


    // warm up
    poly_eval(d_coeffs, d_domain, coeffs_size, domain_size, batch_size, d_results);
    hipDeviceSynchronize();

    hipEventRecord(start, 0);
    poly_eval(d_coeffs, d_domain, coeffs_size, domain_size, batch_size, d_results);
    hipEventRecord(stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, stop);
    printf("runtime : %.3f ms.\n", time);

    // run reference and check correctness

    hipMemcpy(results, d_results, sizeof(test_scalar) * total_results_size, hipMemcpyDeviceToHost);

    for (int i = 0; i < total_results_size; i++)
    {
        std::cout << "Results" << results[i] << std::endl;
    }

    return 0;
}
